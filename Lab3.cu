
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

#define N 27000
#define M 1000

using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

void generateMatrix(int* matrix, int size) {
	srand(time(NULL));
	for (size_t i = 0; i < size; ++i) {
		matrix[i] = rand() % 100;
	}
}

void print(int* matrix, int n, int m) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			cout << matrix[i*m + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void compute(int* matrix, int n, int m, int* result) {
	for (int i = 0; i < n; ++i) {
		result[i] = 0;
		int sz = i * m;
		for (int j = 1; j < m; ++j) {
			if (matrix[sz + j - 1] > matrix[sz + j])
				++result[i];
		}
	}
}

bool checkResult(int* resultCPU, int* resultGPU, int n) {
	for(int i = 0; i < N; ++i){
		if(resultCPU[i] != resultGPU[i])
			return false;
	}
	return true;
}

__global__ void computeCUDA(int* matrix, int n, int m, int* result)
{
	int idxStr = threadIdx.x + blockIdx.x*blockDim.x;
	if(idxStr >= n) return;
	int res = 0;
  int temp0 = matrix[idxStr*m];
	for(int i = 1; i < m; ++i){
    int tempC = matrix[idxStr*m + i];
		if(temp0 > tempC)
			++res;
    temp0 = tempC;
	}
	result[idxStr] = res;
}

int main(void) {
	float elapsedTimeCUDA, elapsedTimeCPU;
	clock_t startCPU;

	int* matrixDEVICE;
	int* resultDEVICE;
	int* resultHOST = new int[N];
	int* matrixHOST = new int[N*M];
	int* resultCPU = new int[N];

	generateMatrix(matrixHOST, N*M);
	startCPU = clock();
	compute(matrixHOST, N, M, resultCPU);
	elapsedTimeCPU = (double)(clock() - startCPU) / CLOCKS_PER_SEC;

	cout << "CPU time = " << elapsedTimeCPU * 1000 << " ms\n";
	cout << "CPU memory throughput = " << N * M * 4 / elapsedTimeCPU / 1024 / 1024 / 1024 << " Gb/s\n";

	hipEvent_t startCUDA, stopCUDA;
	hipEventCreate(&startCUDA);
	hipEventCreate(&stopCUDA);

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	CHECK(hipMalloc(&matrixDEVICE, N * M * 4));
	CHECK(hipMemcpy(matrixDEVICE, matrixHOST, N * M * 4, hipMemcpyHostToDevice));

	CHECK(hipMalloc(&resultDEVICE, N * 4));

	hipEventRecord(startCUDA, 0);
	computeCUDA <<<((N + 255)/256), 256, 40000 >>> (matrixDEVICE, N, M, resultDEVICE);
	hipEventRecord(stopCUDA, 0);
	hipEventSynchronize(stopCUDA);
	CHECK(hipGetLastError());
	hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

	CHECK(hipMemcpy(resultHOST, resultDEVICE, N * 4, hipMemcpyDeviceToHost));

	cout << (checkResult(resultCPU, resultHOST, N) ? "Result is correct" : "Result isn't correct") << endl;

	cout << "CUDA time = " << elapsedTimeCUDA << " ms\n";
	cout << "CUDA memory throughput = " << N * M * 4 / elapsedTimeCUDA / 1024 / 1024 / 1.024 << " Gb/s\n";

	//waitKey(0);
	return 0;
}
