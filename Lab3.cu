
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

#define N 27000
#define M 1000

using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

void generateMatrix(int* matrix, int size) {
	srand(time(NULL));
	for (size_t i = 0; i < size; ++i) {
		matrix[i] = rand() % 100;
	}
}

void print(int* matrix, int n, int m) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			cout << matrix[i*m + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void compute(int* matrix, int n, int m, int* result) {
	for (int i = 0; i < n; ++i) {
		result[i] = 0;
		int sz = i * m;
		for (int j = 1; j < m; ++j) {
			if (matrix[sz + j - 1] > matrix[sz + j])
				++result[i];
		}
	}
}

bool checkResult(int* resultCPU, int* resultGPU, int n) {
	for(int i = 0; i < N; ++i){
		if(resultCPU[i] != resultGPU[i])
			return false;
	}
	return true;
}

__global__ void computeCUDA(int* matrix, int n, int m, int* result)
{
	int idxStr = threadIdx.x + blockIdx.x*blockDim.x;
	if(idxStr >= n) return;
	int res = 0;
	for(int i = 1; i < m; ++i){
		if(matrix[idxStr*m + i - 1] > matrix[idxStr*m + i])
			++res;
	}
	result[idxStr] = res;
}

int main(void) {
	float elapsedTimeCUDA, elapsedTimeCPU;
	clock_t startCPU;

	int* matrixDEVICE;
	int* resultDEVICE;
	int* resultHOST = new int[N];
	int* matrixHOST = new int[N*M];
	int* resultCPU = new int[N];

	generateMatrix(matrixHOST, N*M);
	startCPU = clock();
	compute(matrixHOST, N, M, resultCPU);
	elapsedTimeCPU = (double)(clock() - startCPU) / CLOCKS_PER_SEC;

	cout << "CPU time = " << elapsedTimeCPU * 1000 << " ms\n";
	cout << "CPU memory throughput = " << N * M * 4 / elapsedTimeCPU / 1024 / 1024 / 1024 << " Gb/s\n";

	hipEvent_t startCUDA, stopCUDA;
	hipEventCreate(&startCUDA);
	hipEventCreate(&stopCUDA);

	CHECK(hipMalloc(&matrixDEVICE, N * M * 4));
	CHECK(hipMemcpy(matrixDEVICE, matrixHOST, N * M * 4, hipMemcpyHostToDevice));

	CHECK(hipMalloc(&resultDEVICE, N * 4));

	hipEventRecord(startCUDA, 0);
	computeCUDA <<<((N + 1023)/1024), 1024 >>> (matrixDEVICE, N, M, resultDEVICE);
	hipEventRecord(stopCUDA, 0);
	hipEventSynchronize(stopCUDA);
	CHECK(hipGetLastError());
	hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

	CHECK(hipMemcpy(resultHOST, resultDEVICE, N * 4, hipMemcpyDeviceToHost));

	cout << (checkResult(resultCPU, resultHOST, N) ? "Result is correct" : "Result isn't correct") << endl;

	cout << "CUDA time = " << elapsedTimeCUDA << " ms\n";
	cout << "CUDA memory throughput = " << N * M * 4 / elapsedTimeCUDA / 1024 / 1024 / 1024 << " Gb/s\n";

	//waitKey(0);
	return 0;
}